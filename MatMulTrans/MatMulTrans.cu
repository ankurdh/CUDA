#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>

__global__ void matTransposeMul(int *matrixA, int *matrixB, int *matrixC, int matSize)
{
    int threadCol = blockIdx.x * blockDim.x + threadIdx.x;
    int threadRow = blockIdx.y * blockDim.y + threadIdx.y;

    int k, sum = 0;

    if(threadCol < matSize && threadRow < matSize)
    {
        for(k = 0 ; k < matSize ; k++)
            sum += matrixA[threadRow+matSize*k]*matrixB[k*matSize+threadCol];

    matrixC[threadRow*matSize+threadCol] = sum;
    }
}

void printMatrix(int *matrix, int size, char * matrixName)
{

    if(size > 10)
      return;

    int i = 0;
    printf("Printing Matrix: %s\n", matrixName);
    for( ; i < size * size ; i ++)
    {
        if(i % size == 0)
            printf("\n");

        printf("%-3d  ", matrix[i]);
    }

    printf("\n\n");
}

void checkError(hipError_t error, char * function)
{

        if(error != hipSuccess)
        {
                printf("\"%s\" has a problem with error code %d and desc: %s\n", function, error, hipGetErrorString(error));
                exit(-1);
        }
}

bool checkIfMatricesEqual(int * mat1, int * mat2, int matSize)
{
    int i = 0;
    for( ; i < matSize*matSize; i++)
       if(mat1[i] != mat2[i]){
           printf("values different for i: %d\n", i);
		   printf("mat1[i] = %d, mat2[i] = %d\n", mat1[i], mat2[i]);		   
		   return false;
	   }

    return true;
}

void readValue(int *value, char * msg, int lowerBound, int upperBound)
{
    while(true)
    {
        printf("%s(%d-%d): ", msg, lowerBound, upperBound);
        scanf("%d", value);
        
        if(*value <= upperBound && *value >= lowerBound)
            return;
    }        
}

int main()
{

   //have variables for threads per block, number of blocks.
   int threadsPerBlock = 0, blocksInGrid = 0;

   //create cuda event variables
   hipEvent_t hostStart, hostStop, deviceStart, deviceStop;
   float timeDifferenceOnHost, timeDifferenceOnDevice;

   //program variables
   int matrixSize = 0;
   size_t size;                     //variable to have the size of arrays on device
   int *matA, *matATransposeCPU, *matB, *matC, *matProductGPU;   //matrices for host
   int *gpuMatA, *gpuMatB, *gpuMatC;            //matrices for Device

   //initialize cuda timing variables
   hipEventCreate(&hostStart);
   hipEventCreate(&hostStop);
   hipEventCreate(&deviceStart);
   hipEventCreate(&deviceStop);
  
   printf("Enter the size of the matrix: ");
   scanf("%d", &matrixSize);

   //calculate the size required on GPU
   size = matrixSize * matrixSize * sizeof(int);

   matA = (int *)malloc(matrixSize * sizeof(int) * matrixSize);
   matB = (int *)malloc(matrixSize * sizeof(int) * matrixSize);
   matC = (int *)malloc(matrixSize * sizeof(int) * matrixSize);
   matATransposeCPU = (int *)malloc(matrixSize * sizeof(int) * matrixSize);
   matProductGPU = (int *)malloc(matrixSize * sizeof(int) * matrixSize);

   for(int i = 0 ; i < matrixSize * matrixSize; i ++)
         	  matA[i] = matB[i] = (i*2)%10;

   printMatrix(matA, matrixSize, "Matrix A");
   printMatrix(matB, matrixSize, "Matrix B");
   for(int i = 0 ; i < matrixSize; i++){
	   int sum;
	   for(int j = 0 ; j < matrixSize ; j ++){
		   sum = 0;
           for(int k = 0 ; k < matrixSize ; k ++){
			   sum += matA[i*matrixSize+k]*matB[k*matrixSize+j];
			   //printf("A Index: %d, B Index: %d\n", i*matrixSize+k, k*matrixSize+j);
		   }
	   	   matC[i*matrixSize+j] = sum;
		   //printf("C Index: %d\n", i*matrixSize+j);
	   }
   }
   printMatrix(matC, matrixSize, "Matrix C");
   printf("Transposing matrix on CPU...\n");
   for(int i = 0 ; i < matrixSize ; i ++)
   {
	   for(int j = 0 ; j < matrixSize ; j ++)
	   {
    	   //printf("Src Indx: %d, Dstn Indx: %d\n", i*matrixSize+j, j*matrixSize+i);
           matATransposeCPU[j*matrixSize+i] = matA[i*matrixSize+j];

	   }
   }
   
   printMatrix(matATransposeCPU, matrixSize, "Transpose Matrix");

   printf("Transposing Matrix A finished. Multiplying transposed matrix on CPU..i\n");

   hipEventRecord(hostStart, 0);
   for(int i = 0 ; i < matrixSize; i ++){
	   for(int j = 0 ; j < matrixSize ; j ++){
		   int sum = 0;
		   for(int k = 0 ; k < matrixSize ; k ++)
			   sum += matATransposeCPU[k*matrixSize+i] * matB[k*matrixSize+j];
		   matC[i*matrixSize+j] = sum;

       }
   }

   hipEventRecord(hostStop, 0);
   hipEventElapsedTime(&timeDifferenceOnHost, hostStart, hostStop);
   printf("Matrix transpose mulitpication over. Time taken on CPU: %5.5f\n", timeDifferenceOnHost);

   printMatrix(matC, matrixSize, "Product Matrix");
   //allocate memory on GPU
   checkError(hipMalloc((void**)&gpuMatA, size), "Malloc for Matrix A");
   checkError(hipMalloc((void**)&gpuMatB, size), "Malloc for Matrix B");
   checkError(hipMalloc((void**)&gpuMatC, size), "Malloc for Matrix C");

   //copy the matrix A and matrix B
   checkError(hipMemcpy(gpuMatA, matATransposeCPU, size, hipMemcpyHostToDevice), "Matrix A Copy");
   checkError(hipMemcpy(gpuMatB, matB, size, hipMemcpyHostToDevice), "Matrix B Copy");

   bool done = false;
 
   while(!done)   
   {

	   //create a proper grid block using dim3
	   readValue(&threadsPerBlock, "Enter no. of threads per block(input of 'P' will construct PxP threads in block)", 4, 32);
	   readValue(&blocksInGrid, "Enter no. of blocks in grid(input of 'P' will construct PxP blocks)", (matrixSize + threadsPerBlock -1)/threadsPerBlock, 65535);
	   printf("Threads Per block: %d, Blocks in grid: %d\n", threadsPerBlock, blocksInGrid); 
	   printf("Multiplying matrices on GPU..\n");
	   dim3 blocks(threadsPerBlock, threadsPerBlock);                                                   
	   dim3 grid(blocksInGrid, blocksInGrid); //(matrixSize + threadsPerBlock - 1/blocks.x), (matrixSize + blocks.y - 1/blocks.y));
	
	   //call the kernels to execute
	   hipEventRecord(deviceStart, 0);
	   printf("Total linear threads: %d\n", blocksInGrid*threadsPerBlock);
	   matTransposeMul<<<grid, blocks>>>(gpuMatA, gpuMatB,gpuMatC, matrixSize);
	   hipDeviceSynchronize();
	   hipEventRecord(deviceStop, 0);
	   //hipEventSynchronize(deviceStop);
	
	   hipEventElapsedTime(&timeDifferenceOnDevice, deviceStart, deviceStop);
	
	   //copy the result back into host memory
	   checkError(hipMemcpy(matProductGPU, gpuMatC, size, hipMemcpyDeviceToHost), "Matrix C Copy from device to Host");
	
	   if(checkIfMatricesEqual(matC, matProductGPU, matrixSize)){
	      printf("Kernels correct!\n");
		  if(matrixSize < 10)
			  printMatrix(matProductGPU, matrixSize, "Product matrix from GPU:");
	   }
	   else
	      printf("Kernel logic wrong!\n");
	
	   printf("Finished multiplying transpose matrix on GPU. Time taken: %5.5f\n", timeDifferenceOnDevice);
	   printf("Speedup: %5.5f\n", (float)timeDifferenceOnHost/timeDifferenceOnDevice);
	
	   char c = 'n';
	   printf("Again?(y/n): ");
	   while(true)
	      {
	         c = getchar();
	         if(c == 'y' || c == 'n')
	   	   break;
	      }
	   if(c == 'n')
	     break;
	
   }
     
   free(matA);
   free(matB);
   free(matC);
   free(matProductGPU);
   free(matATransposeCPU);
	
   hipEventDestroy(deviceStart);
   hipEventDestroy(deviceStop);
   hipEventDestroy(hostStart);
   hipEventDestroy(hostStop);
 
   return 0;
   
}
