
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void matTranspose(int *matrixA, int *matrixB, int matSize)
{
    int threadCol = blockIdx.x * blockDim.x + threadIdx.x;
    int threadRow = blockIdx.y * blockDim.y + threadIdx.y;
    
    int srcIndex = threadRow*matSize+threadCol;
	int dstnIndex = threadCol*matSize+threadRow;

	matrixB[dstnIndex] = matrixA[srcIndex];

}

void printMatrix(int *matrix, int size, char * matrixName)
{

    if(size > 10)
      return;

    int i = 0;
    printf("Printing Matrix: %s\n", matrixName);
    for( ; i < size * size ; i ++)
    {
        if(i % size == 0)
            printf("\n");

        printf("%-3d  ", matrix[i]);
    }

    printf("\n\n");
}

void checkError(hipError_t error, char * function)
{

        if(error != hipSuccess)
        {
                printf("\"%s\" has a problem with error code %d and desc: %s\n", function, error, hipGetErrorString(error));
                exit(-1);
        }
}

bool checkIfMatricesEqual(int * mat1, int * mat2, int matSize)
{
    int i = 0;
    for( ; i < matSize; i++)
       if(mat1[i] != mat2[i]){
           printf("values different for i: %d\n", i);
		   printf("mat1[i] = %d, mat2[i] = %d\n", mat1[i], mat2[i]);		   
		   return false;
	   }

    return true;
}

void readValue(int *value, char * msg, int lowerBound, int upperBound)
{
    while(true)
    {
        printf("%s(%d-%d): ", msg, lowerBound, upperBound);
        scanf("%d", value);
        
        if(*value <= upperBound && *value >= lowerBound)
            return;
    }        
}

int main()
{

   //have variables for threads per block, number of blocks.
   int threadsPerBlock = 0, blocksInGrid = 0;

   //create cuda event variables
   hipEvent_t hostStart, hostStop, deviceStart, deviceStop;
   float timeDifferenceOnHost, timeDifferenceOnDevice;

   //program variables
   int matrixSize = 0;
   size_t size;                     //variable to have the size of arrays on device
   int *matA, *matB, *matC, *matCFromGPU;   //matrices for host
   int *gpuMatA, *gpuMatB, *gpuMatC;            //matrices for Device

   //initialize cuda timing variables
   hipEventCreate(&hostStart);
   hipEventCreate(&hostStop);
   hipEventCreate(&deviceStart);
   hipEventCreate(&deviceStop);
  
   printf("Enter the size of the matrix: ");
   scanf("%d", &matrixSize);

   //calculate the size required on GPU
   size = matrixSize * matrixSize * sizeof(int);

   matA = (int *)malloc(matrixSize * sizeof(int) * matrixSize);
   matB = (int *)malloc(matrixSize * sizeof(int) * matrixSize);
   matC = (int *)malloc(matrixSize * sizeof(int) * matrixSize);

   for(int i = 0 ; i < matrixSize * matrixSize; i ++)
         	  matA[i] = (i*2)%10;

   printMatrix(matA, matrixSize, "Matrix A");
   //printMatrix(matB, matrixSize, "Matrix B");
   printf("Transposing matrix on CPU...\n");
   hipEventRecord(hostStart, 0);
   //matrix multiplication code goes here.
   for(int i = 0 ; i < matrixSize ; i ++)
   {
	   for(int j = 0 ; j < matrixSize ; j ++)
	   {
    	   //printf("Src Indx: %d, Dstn Indx: %d\n", i*matrixSize+j, j*matrixSize+i);
           matB[j*matrixSize+i] = matA[i*matrixSize+j];

	   }
   }
   hipEventRecord(hostStop, 0);
   hipEventElapsedTime(&timeDifferenceOnHost, hostStart, hostStop);
   printf("Matrix transpose over. Time taken on CPU: %5.5f\n", timeDifferenceOnHost);
   
   printMatrix(matB, matrixSize, "Transpose Matrix");
   
   //allocate memory on GPU
   checkError(hipMalloc((void**)&gpuMatA, size), "Malloc for Matrix A");
   checkError(hipMalloc((void**)&gpuMatB, size), "Malloc for Matrix B");
   //checkError(cudaMalloc((void**)&gpuMatC, size), "Malloc for Matrix C");

   //copy the matrix A and matrix B
   checkError(hipMemcpy(gpuMatA, matA, size, hipMemcpyHostToDevice), "Matrix A Copy");
   //checkError(cudaMemcpy(gpuMatB, matB, size, cudaMemcpyHostToDevice), "Matrix B Copy");

   bool done = false;
 
   while(!done)   
   {
	   matCFromGPU = (int *)malloc(matrixSize * sizeof(int) * matrixSize);

	   //create a proper grid block using dim3
	   readValue(&threadsPerBlock, "Enter no. of threads per block(input of 'P' will construct PxP threads in block)", 4, 32);
	   readValue(&blocksInGrid, "Enter no. of blocks in grid(input of 'P' will construct PxP blocks)", (matrixSize + threadsPerBlock -1)/threadsPerBlock, 65535);
	   printf("Threads Per block: %d, Blocks in grid: %d\n", threadsPerBlock, blocksInGrid); 
	   printf("Multiplying matrices on GPU..\n");
	   dim3 blocks(threadsPerBlock, threadsPerBlock);                                                   
	   dim3 grid(blocksInGrid, blocksInGrid); //(matrixSize + threadsPerBlock - 1/blocks.x), (matrixSize + blocks.y - 1/blocks.y));
	
	   //call the kernels to execute
	   hipEventRecord(deviceStart, 0);
	   printf("Total linear threads: %d\n", blocksInGrid*threadsPerBlock);
	   matTranspose<<<grid, blocks>>>(gpuMatA, gpuMatB, matrixSize);
	   hipEventRecord(deviceStop, 0);
	   hipEventSynchronize(deviceStop);
	
	   hipEventElapsedTime(&timeDifferenceOnDevice, deviceStart, deviceStop);
	
	   //copy the result back into host memory
	   checkError(hipMemcpy(matCFromGPU, gpuMatB, size, hipMemcpyDeviceToHost), "Matrix C Copy from device to Host");
	
	   if(checkIfMatricesEqual(matB, matCFromGPU, matrixSize))
	      printf("Kernels correct!\n");
	   else
	      printf("Kernel logic wrong!\n");
	
	   printf("Finished transposing on GPU. Time taken: %5.5f\n", timeDifferenceOnDevice);   
	   printf("Speedup: %5.5f\n", (float)timeDifferenceOnHost/timeDifferenceOnDevice);
	
	   printMatrix(matCFromGPU, matrixSize, "Transpose Matrix from GPU");
	
	   char c = 'n';
	   printf("Again?(y/n): ");
	   while(true)
	      {
	         c = getchar();
	         if(c == 'y' || c == 'n')
	   	   break;
	      }
	   if(c == 'n')
	     break;
	
	   free(matCFromGPU);
   }
     
   free(matA);
   free(matB);
   free(matC);
	
   hipEventDestroy(deviceStart);
   hipEventDestroy(deviceStop);
   hipEventDestroy(hostStart);
   hipEventDestroy(hostStop);
 
   return 0;
   
}
