#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>

__device__ int gpuHistogram[10];

__global__ void computeGpuHistogram(int *arr, int noOfElements)
{
    //clear the global gpu Histogram array
	if(blockIdx.x == 0 && threadIdx.x < 10)
		gpuHistogram[threadIdx.x] = 0;

	//force all threads to wait for the first 10 threads 
	__syncthreads();

	//initialize the counter variables. 
	//NOTE: These variables will be allocated to registers.
	int count0 = 0, 
	    count1 = 0,
		count2 = 0,
		count3 = 0,
		count4 = 0,
		count5 = 0,
		count6 = 0,
		count7 = 0,
		count8 = 0,
		count9 = 0;

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int noOfThreads = blockDim.x * gridDim.x;

    while(tid < noOfElements)
	{
		if(arr[tid] == 0)
			++count0;
		else if(arr[tid] == 1)
			++count1;
        else if(arr[tid] == 2)
			++count2;
		else if(arr[tid] == 3)
			++count3;
		else if(arr[tid] == 4)
			++count4;
		else if(arr[tid] == 5)
			++count5;
		else if(arr[tid] == 6)
			++count6;
		else if(arr[tid] == 7)
			++count7;
		else if(arr[tid] == 8)
			++count8;
		else
			++count9;

		tid += noOfThreads;
	}

    //wait for all threads to complete writing into the shared mem.
	__syncthreads();

	//compute the final histogram.
	atomicAdd(&gpuHistogram[0], count0);
	atomicAdd(&gpuHistogram[1], count1);
	atomicAdd(&gpuHistogram[2], count2);
	atomicAdd(&gpuHistogram[3], count3);
	atomicAdd(&gpuHistogram[4], count4);
	atomicAdd(&gpuHistogram[5], count5);
	atomicAdd(&gpuHistogram[6], count6);
	atomicAdd(&gpuHistogram[7], count7);
	atomicAdd(&gpuHistogram[8], count8);
	atomicAdd(&gpuHistogram[9], count9);

}


void checkError(hipError_t error, char * function)
{
	if(error != hipSuccess)
	{
	    printf("\"%s\" has a problem with error code %d and desc: %s\n", function, error, hipGetErrorString(error));
	    exit(-1);
    }
} 

void readValue(int *value, char * msg, int lowerBound, int upperBound)
{
	while(1)
	{
		printf("%s(%d-%d): ", msg, lowerBound, upperBound);
		scanf("%d", value);
		
		if(*value <= upperBound && *value >= lowerBound)
		    return;	

		printf("Incorrect values. Enter again.\n");
	}
}

void fillArrayWithRandNos(int * arr, int noOfElements)
{
	int i;
	srand(5);												//for consistent numbers on every run.
	if(noOfElements < 20)
	{
		for(i = 0 ; i < noOfElements; ++i)
		{
            arr[i] = rand()%10;
			printf("%d   ", arr[i]);
		}
        printf("\n");
		return;
	}

	for(i = 0 ; i < noOfElements; ++i)
		arr[i] = rand()%10;
}

void computeHistogram(int *arr, int *histogram, int noOfElements)
{
	int i;
	for(i = 0 ; i < noOfElements ; ++i)
		++histogram[arr[i]];
}

bool cpuGpuResultsCompare(int *cpuResultsArray, int * histogramFromGPU)
{
	for(int i = 0 ; i < 10 ; i ++)
    	if(cpuResultsArray[i] != histogramFromGPU[i])
			return false;

	return true;
}

int main()
{
	int noOfElements = -1, i;
	int *arr, *gpuArray;
    size_t size; 
	
	//have variables for threads per block, number of blocks.
    int threadsPerBlock = 0, blocksInGrid = 0;

    //create cuda event variables
    hipEvent_t hostStart, hostStop, deviceStart, deviceStop;
    float timeDifferenceOnHost, timeDifferenceOnDevice;
    
    //create cuda events.
    hipEventCreate(&hostStart);
	hipEventCreate(&hostStop);
	hipEventCreate(&deviceStart);
	hipEventCreate(&deviceStop);
    while(1)
	{
		int histogram[10] = {0,0,0,0,0,0,0,0,0,0}, histogramFromGPU[10];
		
		printf("Enter the no. of elements to run test on: ");
		scanf("%d", &noOfElements);

    	arr = (int *)malloc(noOfElements * sizeof(int));

		printf("Filling array with random numbers...\n");
		fillArrayWithRandNos(arr, noOfElements);

	    printf("Computing histogram on CPU...\n");
		hipEventRecord(hostStart, 0);
	    computeHistogram(arr, histogram, noOfElements);
		hipEventRecord(hostStop, 0);
	    hipEventElapsedTime(&timeDifferenceOnHost, hostStart, hostStop);

		//printf("Computation over. Results of CPU computation:\n");
		//for(i = 0 ; i < 10 ; ++i)
	        //printf("No of %d: %d\n", i, histogram[i]);
	
	
		size = noOfElements * sizeof(int);
	    checkError(hipMalloc((void**)&gpuArray, size), "Mallocing array on GPU");
	
		checkError(hipMemcpy(gpuArray, arr, size, hipMemcpyHostToDevice), "Input array copy");
	   
		//create a proper grid block using dim3
	    readValue(&threadsPerBlock, "Enter no. of threads per block(input of 'P' will construct a P threaded linear block)", 4,1024);
		readValue(&blocksInGrid, "Enter no. of blocks in grid(input of 'P' will construct linear grid with P blocks)", 0, 65535/threadsPerBlock+1);
	
		hipEventRecord(deviceStart, 0);
	 	computeGpuHistogram<<<blocksInGrid, threadsPerBlock>>>(gpuArray, noOfElements);   
	    hipDeviceSynchronize();
	    hipEventRecord(deviceStop, 0);
		hipEventElapsedTime(&timeDifferenceOnDevice, deviceStart, deviceStop);
	
		hipMemcpyFromSymbol(&histogramFromGPU,HIP_SYMBOL("gpuHistogram"), sizeof(histogramFromGPU), 0, hipMemcpyDeviceToHost);
	
	  	if(cpuGpuResultsCompare(histogram, histogramFromGPU))
			printf("GPU and CPU results match\n");
		else
			printf("GPU and CPU results don't match\n");

		printf("CPU & GPU stats: \n");
		for(i = 0 ; i < 10 ; ++i)
	        printf("No of %ds: %d %d\n", i, histogram[i], histogramFromGPU[i]);
			
		printf("Time on CPU : %5.5f, Time on GPU: %5.5f\n", timeDifferenceOnHost, timeDifferenceOnDevice);
	
		printf("-----------------------------------------------\n");
		printf("Speedup: %5.5f\n", timeDifferenceOnHost/timeDifferenceOnDevice);
	
    	free(arr);
		hipFree(gpuArray);

		char c = 'n';
	 	printf("Again?(y/n): ");
		while(true)
		{
		   c = getchar();
	       if(c == 'y' || c == 'n')
	   	  	  break;
	    }
	   
	    if(c == 'n')
	       break;
	}	 
	printf("\n");

	hipEventDestroy(deviceStop);
	hipEventDestroy(deviceStart);
	hipEventDestroy(hostStart);
	hipEventDestroy(hostStop);

    return 0;
}
