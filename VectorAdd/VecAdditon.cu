#include<stdio.h>
#include<hip/hip_runtime.h>

#define N 10

__global__ void vecAdd(int *a, int *b, int *c)
{
    int id = blockIdx.x;
    if(id < N)
	c[id] = a[id] + b[id];
}

void checkError(hipError_t error, char * function)
{

	if(error != hipSuccess)
	{
		printf("\"%s\" has a problem with error code %d and desc: %s\n", function, error, hipGetErrorString(error));
		exit(-1);
	}
}

int main()
{
	int a[N], b[N], c[N];
	int *deviceA, *deviceB, *deviceC;
	int i; //a variable for use in loops.
	size_t size = N * sizeof(int);

	//put some value in the 'a' & 'b' arrays
	for(i = 0 ; i < N ; i ++)
	{
		a[i] = i;
		b[i] = i;
	}

	//initialize the memory on GPU
	checkError(hipMalloc((void**)&deviceA, size), "Cuda Malloc for deviceA");
	checkError(hipMalloc((void**)&deviceB, size), "Cuda Malloc for deviceB");
	checkError(hipMalloc((void**)&deviceC, size), "Cuda Malloc for deviceC");

	checkError(hipMemcpy(deviceA, a, size, hipMemcpyHostToDevice), "Cuda MemCpy for DeviceA");
	checkError(hipMemcpy(deviceB, b, size, hipMemcpyHostToDevice), "Cuda MemCpy for DeviceB");	

	vecAdd<<<N , 1>>>(deviceA, deviceB, deviceC);

	checkError(hipMemcpy(c, deviceC, size, hipMemcpyDeviceToHost), "Cuda MemCpy for DeviceC");

	for(i = 0 ; i < N ; i ++)
		printf("c[%d] = %d\n", i , c[i]);

	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);	

	return 0;
}
